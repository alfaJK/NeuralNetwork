﻿#include <time.h> 
#include "hip/hip_runtime.h"

#include "iostream"
#include <stdio.h>

#define BLOCK_SIZE 32

class Unified {
public:
    void* operator new(size_t len) {
        void* ptr;
        hipMallocManaged(&ptr, len);
        return ptr;
    }
    void operator delete(void* ptr) {
        hipFree(ptr);
    }
    void* operator new[](std::size_t size) {
        void* ptr;
        hipMallocManaged(&ptr, size);
        return ptr;
    }
        void operator delete[](void* ptr) {
        hipFree(ptr);
    }
};


class Matrix:public Unified {
private:
    int* Data;
public:
    int height;
    int width;
    __host__  __device__ Matrix(int w, int h, int* data) {
        height = h;
        width = w;
        hipMallocManaged(&Data, width * height * sizeof(int));
        hipMemcpy(Data, data, width * height * sizeof(int), hipMemcpyHostToDevice);
    }
    __host__  __device__ int GetElement(int x,int y);
    __host__  __device__ int* GetSubMatrix(int x, int y);
    __host__  __device__ void SetElement(int x, int y,int data);
    __host__  __device__ void PrintMatrix();
};

__host__ __device__ int Matrix::GetElement(int x, int y) {
    return Data[width * y + x];
}
__host__  __device__ int* Matrix::GetSubMatrix(int x, int y) {

   // Matrix *Asub = new Matrix(BLOCK_SIZE, BLOCK_SIZE,&Data[width * BLOCK_SIZE * y + BLOCK_SIZE * x]);
    return &Data[width * BLOCK_SIZE * y + BLOCK_SIZE * x];
}
__host__ __device__ void Matrix::SetElement(int x, int y, int data) {
    Data[width * y + x] = data;
}

__host__ __device__ void Matrix::PrintMatrix() {
  //  int* Data2 = (int*)malloc(width * height * sizeof(int));
  //  cudaMemcpy(Data2, Data, width * height * sizeof(int), cudaMemcpyDeviceToHost);
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            printf(" %d", Data[width * y + x]);
   //         std::cout << Data2[width * y + x] << " ";
        }
        printf("\n");
    }
   // delete Data2;
}




int H =900;
int W = 900;
int F_W = 17;
int SIZE = W * H;




/*__device__ int CalcElement(int a, int b) {
    return a + b;
}




 void AddMatrix(int* a, int* b, int* c, int ww)
{
     clock_t start = clock();
   
     for (int i = 0; i < ww; i++) {
         for (int j = 0; j < ww; j++) {
             c[ww * i + j] = a[ww * i + j]+ b[ww * i + j];
         }
    }
     clock_t end = clock();
     double seconds = (double)(end - start);
     printf("The time: %f ms\n", seconds);


}


__global__ void AdditionMatrix( int *a,  int *b, int *c,int ww)
{

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ int AResMatrix[BLOCK_SIZE][BLOCK_SIZE];

    AResMatrix[threadIdx.x][threadIdx.y] = a[ww * index_y + index_x]+ b[ww * index_y + index_x];

    __syncthreads();

    c[ww * index_y + index_x] = AResMatrix[threadIdx.x][threadIdx.y];
    __syncthreads();
}

void printMatrix(int* c, int ww) {
    for (int x = 0; x < ww;x++) {
        for (int y = 0; y < ww; y++) {
            std::cout << c[ww * y + x]<<" ";
        }
        std::cout << std::endl;
    }
}
*/




void AddMatrix(Matrix* InputMatrix, Matrix* Filter, Matrix* OutputMatrix)
{
    clock_t start = clock();

    for (int y = 0; y < InputMatrix->height; y++) {
        for (int x = 0; x < InputMatrix->width; x++) {
            int sum = 0;
            int Padding = (Filter->width - 1) / 2;
            for (int i = 0; i < Filter->height; i++) {
                for (int j = 0; j < Filter->width; j++) {
                    int i0 = y + i - Padding;
                    int j0 = x + j - Padding;
                    if (i0 < 0 || i0 >= InputMatrix->height || j0 < 0 || j0 >= InputMatrix->width)
                        continue;
                    sum += InputMatrix->GetElement(i0, j0) * Filter->GetElement(i, j);
                }
            }
            OutputMatrix->SetElement(y, x, sum);
        }
    }

    clock_t end = clock();
    double seconds = (double)(end - start);
    printf("The time: %f ms\n", seconds);
   /* for (int i = 0; i < ww; i++) {
        for (int j = 0; j < ww; j++) {
            c->SetElement(i, j, a->GetElement(i, j) + b->GetElement(i, j));
        }
    }
    clock_t end = clock();
    double seconds = (double)(end - start);
    printf("The time: %f ms\n", seconds);

    */
}

__device__ int calc(int index_x, int index_y , Matrix *input , Matrix* filter) {

    int sum = 0;
    int Padding = (filter->width - 1) / 2;

    for (int i = 0; i < filter->height; i++) {
        for (int j = 0; j < filter->width; j++) {
            int i0 = index_x + i - Padding;
            int j0 = index_y + j - Padding;
            if (i0 < 0 || i0 >= input->height || j0 < 0 || j0 >= input->width)
                continue;
            sum += input->GetElement(i0, j0) * filter->GetElement(i, j);
        }
    }
    return sum;
}

__global__ void NotOptAdditionMatrix(Matrix* InputMatrix, Matrix* Filter, Matrix* OutputMatrix )
{
   // int block_x = blockIdx.x;
   // int block_y = blockIdx.y;

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;


    int sum = calc(index_x, index_y, InputMatrix, Filter);
    __syncthreads();

    OutputMatrix->SetElement(index_x, index_y, sum);


    __syncthreads();



    // cudaDeviceSynchronize();
   // c->SetElement(index_x , index_y, a->GetElement(index_x, index_y)+ b->GetElement(index_x, index_y));

}


__global__ void AdditionMatrix(Matrix* InputMatrix, Matrix* Filter, Matrix* OutputMatrix)
{

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int sum = 0;
    int Padding = (Filter->width - 1) / 2;
    int* OutSubM = OutputMatrix->GetSubMatrix(blockIdx.x, blockIdx.y);

    for (int m = 0; m < (InputMatrix->width / BLOCK_SIZE); ++m) {

        int* SubM = InputMatrix->GetSubMatrix(m, blockIdx.y);
        __shared__ int InputSubMatrix[BLOCK_SIZE][BLOCK_SIZE];
        InputSubMatrix[threadIdx.x][threadIdx.y] = SubM[BLOCK_SIZE * threadIdx.y + threadIdx.x];

        __syncthreads();

        for (int i = 0; i < Filter->height; i++) {
            for (int j = 0; j < Filter->width; j++) {
                int i0 = index_x + i - Padding;
                int j0 = index_y + j - Padding;
                if (i0 < 0 || i0 >= BLOCK_SIZE || j0 < 0 || j0 >= BLOCK_SIZE)
                    continue;
                sum += InputSubMatrix[i0][j0] * Filter->GetElement(i, j);
            }
           // __syncthreads();
        }

        __syncthreads();
    
  
    
    }

    OutSubM[BLOCK_SIZE * threadIdx.y + threadIdx.x] = sum;




}

__host__ int main()
{
    int* h_A = (int*)malloc(SIZE * sizeof(int));
    int* h_filter = (int*)malloc(F_W * F_W * sizeof(int));
    int* h_C = (int*)malloc(SIZE * sizeof(int));

    for (int i = 0; i < F_W; i++) {
        for (int j = 0; j < F_W; j++) {
            h_filter[i * F_W + j] =1;
        }
    }


    for (int i = 0; i < W; i++) {
        for (int j = 0; j < H; j++) {
            h_A[i * W + j] = 1;
        }
    }

    Matrix* InputMatrix = new Matrix(W, W, h_A);
  //  InputMatrix->PrintMatrix();
    Matrix* Filter = new Matrix(F_W, F_W, h_filter);
   // Filter->PrintMatrix();
    Matrix* OutputMatrix = new Matrix(W, W, h_A);

    AddMatrix(InputMatrix, Filter, OutputMatrix);

   // OutputMatrix->PrintMatrix();


    dim3 gridSize = dim3(W / BLOCK_SIZE, W / BLOCK_SIZE, 1);
    dim3 blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);




    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

 /*   cudaEventRecord(start);
    AdditionMatrix << <gridSize, blockSize >>> (InputMatrix, Filter, OutputMatrix);
  //  NotOptAdditionMatrix <<<gridSize, blockSize >>> (InputMatrix, Filter, OutputMatrix);
    cudaEventRecord(stop);
    cudaDeviceSynchronize(); cudaGetLastError();
    float milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    std::cout << "CUDA  fast time simple (ms): " << milliseconds << std::endl;
    OutputMatrix->PrintMatrix();
    */

    hipEventRecord(start);
      NotOptAdditionMatrix <<<gridSize, blockSize >>> (InputMatrix, Filter, OutputMatrix);
    //AdditionMatrix << <gridSize, blockSize >>> (InputMatrix, Filter, OutputMatrix);
    hipEventRecord(stop);
    hipDeviceSynchronize(); hipGetLastError();
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "CUDA not fast time simple (ms): " << milliseconds << std::endl;
  //  OutputMatrix->PrintMatrix();




  /*  cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);

    cudaEventRecord(start);
    NotOptAdditionMatrix <<<gridSize, blockSize >>> (A, B, C, W);
    cudaEventRecord(stop);
    cudaDeviceSynchronize(); cudaGetLastError();
    float milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    std::cout << "CUDA not fast time simple (ms): " << milliseconds << std::endl;

    cudaEventRecord(start);
    AdditionMatrix <<<gridSize, blockSize >>> (A, B, C, W);
    cudaEventRecord(stop);
    cudaDeviceSynchronize(); cudaGetLastError();
    milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    std::cout << "CUDA fast time simple (ms): " << milliseconds << std::endl;
    */



  //  C->PrintMatrix();
  //  C->PrintMatrix();
    






  //  AddMatrix(h_A, h_B, h_C, W);

  /*  int* MatrixA;
    int* MatrixB;
    int* MatrixC;
    float milliseconds = 0;
    cudaMalloc(&MatrixA, SIZE * sizeof(int));
    cudaMalloc(&MatrixB, SIZE * sizeof(int));
    cudaMalloc(&MatrixC, SIZE * sizeof(int));

    cudaMemcpy(MatrixA, h_A, SIZE * sizeof(int), cudaMemcpyHostToDevice);
    cudaMemcpy(MatrixB, h_B, SIZE * sizeof(int), cudaMemcpyHostToDevice);

    cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);

    dim3 gridSize = dim3(W / BLOCK_SIZE, W / BLOCK_SIZE, 1);
    dim3 blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

    cudaEventRecord(start);
        AdditionMatrix <<<gridSize, blockSize >>>(MatrixA, MatrixB, MatrixC,W);
    cudaEventRecord(stop);
    cudaDeviceSynchronize(); cudaGetLastError();
    milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    std::cout << "CUDA fast time simple (ms): " << milliseconds << std::endl;
    
    cudaEventRecord(start);
    NotOptAdditionMatrix <<<gridSize, blockSize >>> (MatrixA, MatrixB, MatrixC, W);
    cudaEventRecord(stop);
    cudaDeviceSynchronize(); cudaGetLastError();
    milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    std::cout << "CUDA not fast time simple (ms): " << milliseconds << std::endl;
    cudaMemcpy(h_C, MatrixC, SIZE * sizeof(int), cudaMemcpyDeviceToHost);
    printMatrix(h_C, W);*/
    return 0;
}
