#include "hip/hip_runtime.h"
#include "CudaConv.h"

__device__ int d_ConvDirectDist(Matrix& InputMatrix , Matrix& Filter){

    int sum = 0;
    int Padding = (Filter.width - 1) / 2;

    for (int i = 0; i < Filter.height; i++) {
        for (int j = 0; j < Filter.width; j++) {
            int i0 = index_x + i - Padding;
            int j0 = index_y + j - Padding;
            if (i0 < 0 || i0 >= InputMatrix.height || j0 < 0 || j0 >= InputMatrix.width) continue;
            sum += InputMatrix.GetElement(i0, j0) * Filter.GetElement(i, j);
        }
    }
    return sum;

}

__global__ void ConvDirectDist_global(Matrix& InputMatrix , Matrix& Filter , Matrix& OutputMatrix){

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int sum = d_ConvDirectDist(index_x, index_y, InputMatrix, Filter);
    __syncthreads();

    // ========== РЕАЛИЗОВАТЬ ПЕРЕНОС ДАННЫХ В ВЫХОДНУЮ МАТРИЦУ ==========

   // OutputMatrix->SetElement(index_x, index_y, sum);


    __syncthreads();
}

__device__ void d_ConvBackDist(Matrix& InputMatrix , Matrix& Filter , Matrix& OutputMatrix){



}

__global__ void ConvBackDist_global(Matrix& InputMatrix , Matrix& Filter , Matrix& OutputMatrix){


}

DECLSPEC Matrix CudaConvDirectDist(Matrix& InputMatrix){
    
    dim3 gridSize = dim3(W / BLOCK_SIZE, W / BLOCK_SIZE, 1);
    dim3 blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    ConvDirectDist_global <<<gridSize, blockSize >>> (InputMatrix, OutputMatrix);
    hipEventRecord(stop);

 //   hipDeviceSynchronize(); hipGetLastError();

}

DECLSPEC Matrix CudaConvBackDist(Matrix& InputMatrix , Matrix& Filter){

    dim3 gridSize = dim3(W / BLOCK_SIZE, W / BLOCK_SIZE, 1);
    dim3 blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    ConvBackDist_global <<<gridSize, blockSize >>> (InputMatrix, Filter, OutputMatrix);
    hipEventRecord(stop);

}