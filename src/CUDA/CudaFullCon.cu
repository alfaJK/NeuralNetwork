#include "hip/hip_runtime.h"
#include "CudaFullCon.h"

__device__ void d_FullConDirectDist();

__device__ void d_FullConBackDist();

__global__ void FullConDirectDist_global();

__global__ void FullConBackDist_global();

DECLSPEC Matrix CudaFullConDirectDist();

DECLSPEC Matrix CudaFullConBackDist();