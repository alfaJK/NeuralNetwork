#include "hip/hip_runtime.h"
#include "CudaConv.h"

__device__ void d_ConvDirectDist(Matrix& InputMatrix , Matrix& OutputMatrix){



}

__global__ void ConvDirectDist_global(Matrix& InputMatrix , Matrix& OutputMatrix){

}

__device__ void d_ConvBackDist(Matrix& InputMatrix , Matrix& Filter , Matrix& OutputMatrix){

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int sum = 0;
    int Padding = (Filter->width - 1) / 2;
    int* OutSubM = OutputMatrix->GetSubMatrix(blockIdx.x, blockIdx.y);

    for (int m = 0; m < (InputMatrix->width / BLOCK_SIZE); ++m) {

        int* SubM = InputMatrix->GetSubMatrix(m, blockIdx.y);
        __shared__ int InputSubMatrix[BLOCK_SIZE][BLOCK_SIZE];
        InputSubMatrix[threadIdx.x][threadIdx.y] = SubM[BLOCK_SIZE * threadIdx.y + threadIdx.x];

        __syncthreads();

        for (int i = 0; i < Filter->height; i++) {
            for (int j = 0; j < Filter->width; j++) {
                int i0 = index_x + i - Padding;
                int j0 = index_y + j - Padding;
                if (i0 < 0 || i0 >= BLOCK_SIZE || j0 < 0 || j0 >= BLOCK_SIZE)
                    continue;
                sum += InputSubMatrix[i0][j0] * Filter->GetElement(i, j);
            }
           // __syncthreads();
        }

        __syncthreads();

}

__global__ void ConvBackDist_global(Matrix& InputMatrix , Matrix& Filter , Matrix& OutputMatrix){

}

DECLSPEC Matrix CudaConvDirectDist(Matrix& InputMatrix){
    
    dim3 gridSize = dim3(W / BLOCK_SIZE, W / BLOCK_SIZE, 1);
    dim3 blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    ConvDirectDist_global <<<gridSize, blockSize >>> (InputMatrix, OutputMatrix);
    hipEventRecord(stop);

 //   hipDeviceSynchronize(); hipGetLastError();

}

DECLSPEC Matrix CudaConvBackDist(Matrix& InputMatrix , Matrix& Filter){

    dim3 gridSize = dim3(W / BLOCK_SIZE, W / BLOCK_SIZE, 1);
    dim3 blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    ConvBackDist_global <<<gridSize, blockSize >>> (InputMatrix, Filter, OutputMatrix);
    hipEventRecord(stop);

}