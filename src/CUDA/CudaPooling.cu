#include "hip/hip_runtime.h"
#include "CudaPooling.h"

__device__ void d_PoolingDirectDist();

__device__ void d_PoolingBackDist();

__global__ void PoolingDirectDist_global();

__global__ void PoolingBackDist_global();

DECLSPEC Matrix CudaPoolingDirectDist();

DECLSPEC Matrix CudaPoolingBackDist();