#include "hip/hip_runtime.h"
#include "CudaActivation.h"

__device__ void d_ActivationDirectDist();

__device__ void d_ActivationBackDist();

__global__ void ActivationDirectDist_global();

__global__ void ActivationBackDist_global();

DECLSPEC Matrix CudaActivationDirectDist();

DECLSPEC Matrix CudaActivationBackDist();